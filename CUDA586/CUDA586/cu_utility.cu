#include "hip/hip_runtime.h"
#include "cu_utility.cuh"

// Device Kernels

__device__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

__device__ void matMulVec(const float* W, const float* X, float* Y, int M,
    int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M) {
        float tmp = 0.0;
        for (int k = 0; k < N; k++) {
            tmp += W[i * N + k] * X[k];
        }
        Y[i] = tmp;
    }
}

__device__ float sigmoid(float a) { return 1.0 / (1.0 + exp(-a)); }

__device__ void sigmoid(float* A, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = sigmoid(A[i]);
    }
}

__device__ float d_sigmoid(float a) {
    float xp = exp(-a);
    return xp / ((1.0 + xp) * (1.0 + xp));
}

__device__ void d_sigmoid(float* A, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = d_sigmoid(A[i]);
    }
}

// Global Kernels
__global__ void global_vectorAdd(const float* A, const float* B, float* C,
    int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

__global__ void global_matMulVec(const float* W, const float* X, float* Y,
    int M, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M) {
        float tmp = 0.0;
        for (int k = 0; k < N; k++) {
            tmp += W[i * N + k] * X[k];
        }
        Y[i] = tmp;
    }
}

__global__ void global_sigmoid(float* A, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = sigmoid(A[i]);
    }
}

__global__ void global_d_sigmoid(float* A, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = d_sigmoid(A[i]);
    }
}

__global__ void global_forwardLayer(const float* W, const float* b,
    const float* A, float* result, int M,
    int N) {
    // multiply
    matMulVec(W, A, result, M, N);
    // add
    vectorAdd(result, b, result, N);
    // activate
    sigmoid(result, M);
}

cu_utility::cu_utility(/* args */) {}

cu_utility::~cu_utility() {}

std::vector<float>& cu_utility::cuVectorAdd(const std::vector<float>& x,
    const std::vector<float>& b,
    std::vector<float>& result) {
    if (!(x.size() == b.size() && x.size() == result.size())) {
        std::cerr << "cuVectorAdd - Size does not match!";
        return result;
    }

    int N = x.size();  // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float* d_x, * d_b, * d_r;
    hipMalloc(&d_x, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_r, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    global_vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_x, d_b, d_r, N);

    // Copy result from device to host
    hipMemcpy(result.data(), d_r, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_r);

    return result;
}

std::vector<float>& cu_utility::cuSigmoid(std::vector<float>& x) {
    int N = x.size();  // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float* d_x;
    hipMalloc(&d_x, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    global_sigmoid << <blocksPerGrid, threadsPerBlock >> > (d_x, N);

    // Copy result from device to host
    hipMemcpy(x.data(), d_x, size, hipMemcpyDeviceToHost);

    hipFree(d_x);

    return x;
}

std::vector<float>& cu_utility::cuDSigmoid(std::vector<float>& x) {
    int N = x.size();  // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float* d_x;
    hipMalloc(&d_x, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    global_d_sigmoid << <blocksPerGrid, threadsPerBlock >> > (d_x, N);

    // Copy result from device to host
    hipMemcpy(x.data(), d_x, size, hipMemcpyDeviceToHost);

    hipFree(d_x);

    return x;
}

std::vector<float>& cu_utility::cuMatMulVector(
    const std::vector<std::vector<float>>& W, const std::vector<float>& x,
    std::vector<float>& result) {
    // Check Dims
    if (!(W[0].size() == x.size() && W.size() == result.size())) {
        std::cerr << "cuMatMulVector - Size does not match!";
        return result;
    }

    int M = result.size();
    int N = x.size();  // Size of vectors

    size_t sizeW = M * N * sizeof(float);
    size_t sizeX = N * sizeof(float);
    size_t sizeY = M * sizeof(float);

    // Allocate device memory
    float* d_W, * d_x, * d_y;
    hipMalloc(&d_W, sizeW);
    hipMalloc(&d_x, sizeX);
    hipMalloc(&d_y, sizeY);

    // Copy data from host to device
    std::vector<float> W_flattened(M * N);
    for (int i = 0; i < M; i++) {
        std::copy(W[i].begin(), W[i].end(), W_flattened.begin() + i * N);
    }
    hipMemcpy(d_W, W_flattened.data(), sizeW, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x.data(), sizeX, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    global_matMulVec << <blocksPerGrid, threadsPerBlock >> > (d_W, d_x, d_y, M, N);

    // Copy result from device to host
    hipMemcpy(result.data(), d_y, sizeY, hipMemcpyDeviceToHost);

    hipFree(d_W);
    hipFree(d_x);
    hipFree(d_y);

    return result;
}

std::vector<float>& cu_utility::cuForwardLayer(
    const std::vector<std::vector<float>>& W, const std::vector<float>& b,
    const std::vector<float>& x, std::vector<float>& result) {
    int M = result.size();
    int N = x.size();

    size_t sizeW = M * N * sizeof(float);
    size_t sizeb = M * sizeof(float);
    size_t sizeX = N * sizeof(float);
    size_t sizeY = M * sizeof(float);

    // Allocate device memory
    float* d_W, * d_b, * d_x, * d_y;
    hipMalloc(&d_W, sizeW);
    hipMalloc(&d_b, sizeb);
    hipMalloc(&d_x, sizeX);
    hipMalloc(&d_y, sizeY);

    // Copy data from host to device
    std::vector<float> W_flattened(M * N);
    for (int i = 0; i < M; i++) {
        std::copy(W[i].begin(), W[i].end(), W_flattened.begin() + i * N);
    }
    hipMemcpy(d_W, W_flattened.data(), sizeW, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeb, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x.data(), sizeX, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    global_forwardLayer << <blocksPerGrid, threadsPerBlock >> > (d_W, d_b, d_x, d_y,
        M, N);

    // Copy result from device to host
    hipMemcpy(result.data(), d_y, sizeY, hipMemcpyDeviceToHost);

    hipFree(d_W);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_y);

    return result;
}
