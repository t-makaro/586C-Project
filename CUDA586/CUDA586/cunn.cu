#include "hip/hip_runtime.h"
#include "cunn.cuh"

CUNN::CUNN(std::vector<int> layers) : layers(layers) {
    numLayers = layers.size();
    weights.reserve(numLayers - 1);
    biases.reserve(numLayers - 1);
    activations.reserve(numLayers);

    /*d_weights.reserve(numLayers - 1);
    d_biases.reserve(numLayers - 1);
    d_activations.reserve(numLayers);

    */

    // initialize device pointer vecotrs
	d_weights.resize(numLayers - 1);
	d_biases.resize(numLayers - 1);
	d_activations.resize(numLayers);

	d_activations_batch.resize(numLayers);
}

CUNN::~CUNN() {}

// alloc device weights, biases, activations
void CUNN::deviceAlloc() {
    size_t sizeA0 = layers[0] * sizeof(float); // input vector
    hipMalloc(&d_activations[0], sizeA0);

    for (int i = 1; i < numLayers; i++) {
        int M = layers[i-1];
        int N = layers[i];
        
        size_t sizeWi = M * N * sizeof(float);
        size_t sizeAi = N * sizeof(float);
        size_t sizeBi = N * sizeof(float);

		hipMalloc(&d_weights[i - 1], sizeWi);
		hipMalloc(&d_biases[i - 1], sizeBi);
		hipMalloc(&d_activations[i], sizeAi);
    }
}

void CUNN::deviceFree() {
    hipFree(d_activations[0]);
    for (int i = 1; i < numLayers; i++) {
        hipFree(d_weights[i - 1]);
        hipFree(d_biases[i - 1]);
        hipFree(d_activations[i]);
    }
}

void CUNN::setBatchSizeDevice(int batchSize) {
	this->batchSize = batchSize;

    size_t sizeA0 = layers[0] * batchSize * sizeof(float); // input batch 
    hipMalloc(&d_activations_batch[0], sizeA0);

    for (int i = 1; i < numLayers; i++) {
        int M = layers[i-1];
        int N = layers[i];
        
        size_t sizeAi = N * sizeof(float) * batchSize;
		hipMalloc(&d_activations_batch[i], sizeAi);
    }
}


void CUNN::copyParametersToDevice() {
    deviceAlloc();

    for (int i = 0; i < weights.size(); i++) {
        int M = weights[i].size();
        int N = weights[i][0].size();
        assert(M == biases[i].size());

        size_t sizeWi = M * N * sizeof(float);
        std::vector<float> Wi_flattened(M * N);
        for (int j = 0; j < M; j++) {
            std::copy(weights[i][j].begin(), weights[i][j].end(), Wi_flattened.begin() + j * N);
        }

        hipMemcpy(d_weights[i], Wi_flattened.data(), sizeWi, hipMemcpyHostToDevice);

        size_t sizeBi = M * sizeof(float);
        hipMemcpy(d_biases[i], biases[i].data(), sizeBi, hipMemcpyHostToDevice);
    }
}

void CUNN::testForwardZ(bool isGpu)
{
    zs.reserve(numLayers);
    for (int i = 0; i < numLayers; i++) {
        activations.push_back(Vector(layers[i], 0.0));
        zs.push_back(Vector(layers[i], 0.0));
        if (i < numLayers - 1) {
            dWeights.push_back(Matrix(layers[i + 1], Vector(layers[i], 0.0)));
            dBiases.push_back(Vector(layers[i + 1], 0.0));
        }
    }

    int i = 1;
    if(isGpu)
    {
        cu_utility::cuForwardLayerWithZs(weights[i - 1], biases[i - 1], activations[i - 1], zs[i], activations[i]);
    }
    else
    {
	    forwardZ(weights[i - 1], biases[i - 1], activations[i - 1], zs[i]);
        sigmoid(zs[i], activations[i]);
    }
    std::cout << zs[i][20]; // Breakpoint here to see
}

void CUNN::copyWeights(const std::vector<Matrix> weights) {
    assert(weights.size() == numLayers - 1);
    for (int i = 0; i < weights.size(); i++) {
        assert(weights[i].size() == layers[i + 1]);
        assert(weights[i][0].size() == layers[i]);
    }
    this->weights = weights;
}

void CUNN::copyBiases(const std::vector<Vector> biases) {
    assert(biases.size() == numLayers - 1);
    for (int i = 0; i < biases.size(); i++) {
        assert(biases[i].size() == layers[i + 1]);
    }
    this->biases = biases;
}

float CUNN::sigmoid(float x) { return 1.0 / (1.0 + exp(-x)); }
float CUNN::d_sigmoid(float x) {
    float xp = exp(-x);
    return xp / ((1.0 + xp) * (1.0 + xp));
}

Vector& CUNN::forward(const Vector& x, Vector& result) {
    activations[0] = x;
    for (int i = 1; i < numLayers; i++) {
        forwardLayer(weights[i - 1], biases[i - 1], activations[i - 1],
            activations[i]);
    }
    result = activations[numLayers - 1];
    return result;
}

Vector& CUNN::forwardLayer(const Matrix& w, const Vector& b, const Vector& a,
    Vector& result) {
    // return sigmoid(add(multiply(w, a, result), b, result));
    return cu_utility::cuForwardLayer(w, b, a, result);
}

void CUNN::train(const float* d_trainingData, const int* d_trainingLabels, 
    const int M, const int N, const int iterations, const int batchSize,
    float learningRate) {
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < M; i += batchSize) {
            updateFromBatch(d_trainingData+i*N, d_trainingLabels+i, batchSize, N, learningRate);
        }
    }
}

std::vector<float*> CUNN::allocate_like_weights() {
    std::vector<float*> d_Weights;

    // Allocate zeros to accumulate the gradiant over the batch.
    for (int i = 0; i < numLayers - 1; i++) {
        // Allocate memory on the GPU for change in weights
        float* temp_weights;

        size_t weightSize = layers[i + 1] * layers[i] * sizeof(float);

        // Allocate GPU memory
        hipMalloc(&temp_weights, weightSize);

        // Initialize the allocated memory to 0.0 (optional, but often needed)
        hipMemset(temp_weights, 0.0, weightSize);

        // Store pointers in vectors
        d_Weights.push_back(temp_weights);
    }
    return d_Weights;
}
std::vector<float*> CUNN::allocate_like_biases() {
    std::vector<float*> d_Biases;

    // Allocate zeros to accumulate the gradiant over the batch.
    for (int i = 0; i < numLayers - 1; i++) {
        // Allocate memory on the GPU for change in weights and biases
        float* temp_biases;

        size_t biasSize = layers[i + 1] * sizeof(float);

        // Allocate GPU memory
        hipMalloc(&temp_biases, biasSize);

        // Initialize the allocated memory to 0.0 (optional, but often needed)
        hipMemset(temp_biases, 0.0, biasSize);

        // Store pointers in vectors
        d_Biases.push_back(temp_biases);
    }
    return d_Biases;
}
void deallocateVector(std::vector<float*> vec) {
    for (int i = 0; i < vec.size(); i++) {
        hipFree(vec[0]);
    }
}

void CUNN::updateFromBatch(const float* d_batch, const int* d_labels, 
    const int batchSize, const int N, const float learningRate) {

    std::vector<float*> d_ddWeights = allocate_like_weights();
    std::vector<float*> d_ddBiases = allocate_like_biases();

    // calculate individual gradiants and average them together
    for (int i = 0; i < batchSize; i++) {
        std::vector<float*> d_dWeights = allocate_like_weights();
        std::vector<float*> d_dBiases = allocate_like_biases();
        
        backwards(d_dWeights, d_dBiases, d_batch+i*N, d_labels+i);
        for (int j = 0; j < numLayers-1; j++) {
            cu_utility::d_VectorAdd(d_ddWeights[j], d_dWeights[j], d_ddWeights[j], layers[j + 1] * layers[j], 1.0 / batchSize);
            cu_utility::d_VectorAdd(d_ddBiases[j], d_dBiases[j], d_ddBiases[j], layers[j + 1], 1.0 / batchSize);
        }
        deallocateVector(d_dWeights);
        deallocateVector(d_dBiases);
    }
    // update the weights and biases with gradient computed above at the learning rate
    for (int j = 0; j < numLayers-1; j++) {
        cu_utility::d_VectorAdd(d_weights[j], d_ddWeights[j], d_weights[j], layers[j + 1] * layers[j], -learningRate);
        cu_utility::d_VectorAdd(d_biases[j], d_ddBiases[j], d_biases[j], layers[j + 1], -learningRate);
    }

    deallocateVector(d_ddWeights);
    deallocateVector(d_ddBiases);
}

void CUNN::backwards(std::vector<float*> dWeights_output,
    std::vector<float*> dBiases_output,
    const float* testData, const int* testLabel) {
    // activations[0] = testData;
    for (int i = 1; i < numLayers; i++) {
        forwardZ(weights[i - 1], biases[i - 1], activations[i - 1], zs[i]);
        sigmoid(zs[i], activations[i]);
    }
    Vector delta;
    for (int i = 0; i < numLayers - 1; i++) {
        if (i == 0) {

        }
        else {

        }
    }
}

void CUNN::cost_derivative(const Vector& last_activation, const int label,
    Vector& result) {
    for (int i = 0; i < 10; i++) {
        if (i == label) {
            result[i] = -1 / last_activation[i];
        }
        else {
            result[i] = 1 / (1 - last_activation[i]);
        }
    }
    return;
}

Matrix& CUNN::outer_product(const Vector& a, const Vector& b, Matrix& result) {
    for (int i = 0; i < a.size(); i++) {
        for (int j = 0; j < b.size(); j++) {
            result[i][j] = a[i] * b[j];
        }
    }
    return result;
}

void CUNN::activation_derivative(const Matrix& weights, Vector& z,
    Vector& previous) {
    d_sigmoid(z);
    multiply_elementwise(z, previous, previous);
    Matrix temp;
    transpose(weights, temp);
    Vector result;
    multiply(temp, previous, result);
    previous = result;
}

void CUNN::transpose(const Matrix& a, Matrix& result) {
    // Get the dimensions of the input matrix 'a'
    int rows = a.size();
    int cols = a[0].size();

    // Resize the result matrix to hold the transpose (cols x rows)
    result.resize(cols);
    for (int i = 0; i < cols; ++i) {
        result[i].resize(rows);
    }

    // Perform the transpose operation
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            result[j][i] = a[i][j];
        }
    }
}

float CUNN::evaluate(const Matrix& testData,
    const std::vector<int>& testLabels) {
    int numCorrect = 0;

    // timing
    auto start = std::chrono::high_resolution_clock::now();

    // should be testLabels.size() x 10
    // reserve memory for predictions
	Matrix predictions(testLabels.size(), Vector(10, 0));
	cu_utility::cuForward(d_weights, d_biases, d_activations, layers, testData, predictions);

    for (int i = 0; i < predictions.size(); i++) {
        Vector pred = predictions[i];
        int maxIndex = 0;
        float maxVal = 0;
        for (int j = 0; j < pred.size(); j++) {
            if (pred[j] > maxVal) {
                maxVal = pred[j];
                maxIndex = j;
            }
        }
        if (maxIndex == testLabels[i]) {
            numCorrect++;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    std::cout << "done." << std::endl;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds."
        << std::endl;
    float accuracy = (float)numCorrect / testData.size();
    std::cout << "Train Accuracy: " << accuracy << std::endl;
    return accuracy;
}

float CUNN::evaluate(const float* testData, const std::vector<int>& testLabels) {
    int numCorrect = 0;

    // timing
    auto start = std::chrono::high_resolution_clock::now();
	Matrix predictions(testLabels.size(), Vector(10, 0));
	cu_utility::cuForwardBatch(d_weights, d_biases, d_activations_batch, layers, testData, batchSize, predictions);

    for (int i = 0; i < predictions.size(); i++) {
        Vector pred = predictions[i];
        int maxIndex = 0;
        float maxVal = 0;
        for (int j = 0; j < pred.size(); j++) {
            if (pred[j] > maxVal) {
                maxVal = pred[j];
                maxIndex = j;
            }
        }
        if (maxIndex == testLabels[i]) {
            numCorrect++;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    std::cout << "done." << std::endl;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds."
        << std::endl;
    float accuracy = (float)numCorrect / testLabels.size();
    std::cout << "Train Accuracy: " << accuracy << std::endl;
    return accuracy;

}


Vector& CUNN::forwardZ(const Matrix& w, const Vector& b, const Vector& a,
    Vector& result) {
    return add(multiply(w, a, result), b, result);
}

Vector& CUNN::multiply(const Matrix& w, const Vector& x, Vector& result) {
    cu_utility::cuMatMulVector(w, x, result);
    return result;
}

Vector& CUNN::multiply_elementwise(const Vector& a, const Vector& b,
    Vector& result) {
    for (int i = 0; i < a.size(); i++) {
        result[i] = a[i] * b[i];
    }
    return result;
}

Vector& CUNN::add(const Vector& x, const Vector& b, Vector& result) {
    return cu_utility::cuVectorAdd(x, b, result);
}

Vector& CUNN::add(const Vector& x, const Vector& b, Vector& result,
    const float scale) {
    assert(x.size() == b.size() && x.size() == result.size());

    for (int i = 0; i < x.size(); i++) {
        result[i] = x[i] + b[i] * scale;
    }
    return result;
}

Matrix& CUNN::add(const Matrix& x, const Matrix& b, Matrix& result,
    const float scale) {
    assert(x.size() == b.size() && x.size() == result.size());

    for (int i = 0; i < x.size(); i++) {
        for (int j = 0; j < x[0].size(); j++) {
            result[i][j] = x[i][j] + b[i][j] * scale;
        }
    }
    return result;
}

Vector& CUNN::sigmoid(const Vector& x, Vector& result) {
    result.assign(x.begin(), x.end());
    return cu_utility::cuSigmoid(result);
}

Vector& CUNN::sigmoid(Vector& x) { return cu_utility::cuSigmoid(x); }

Vector& CUNN::d_sigmoid(Vector& x) { return cu_utility::cuDSigmoid(x); }

