#include "hip/hip_runtime.h"
#include "cunn.cuh"

CUNN::CUNN(std::vector<int> layers) : layers(layers) {
    numLayers = layers.size();
    weights.reserve(numLayers - 1);
    biases.reserve(numLayers - 1);
    activations.reserve(numLayers);

    /*d_weights.reserve(numLayers - 1);
    d_biases.reserve(numLayers - 1);
    d_activations.reserve(numLayers);

    */

    // initialize device pointer vecotrs
	d_weights.resize(numLayers - 1);
	d_biases.resize(numLayers - 1);
	d_activations.resize(numLayers);
}

CUNN::~CUNN() {}

// alloc device weights, biases, activations
void CUNN::deviceAlloc() {
    size_t sizeA0 = layers[0] * sizeof(float); // input vector
    hipMalloc(&d_activations[0], sizeA0);

    for (int i = 1; i < numLayers; i++) {
        int M = layers[i-1];
        int N = layers[i];
        
        size_t sizeWi = M * N * sizeof(float);
        size_t sizeAi = N * sizeof(float);
        size_t sizeBi = N * sizeof(float);

		hipMalloc(&d_weights[i - 1], sizeWi);
		hipMalloc(&d_biases[i - 1], sizeBi);
		hipMalloc(&d_activations[i], sizeAi);
    }
}

void CUNN::copyParametersToDevice() {
    deviceAlloc();

    for (int i = 0; i < weights.size(); i++) {
        int M = weights[i].size();
        int N = weights[i][0].size();
        assert(M == biases[i].size());

        size_t sizeWi = M * N * sizeof(float);
        std::vector<float> Wi_flattened(M * N);
        for (int j = 0; j < M; j++) {
            std::copy(weights[i][j].begin(), weights[i][j].end(), Wi_flattened.begin() + j * N);
        }

        hipMemcpy(d_weights[i], Wi_flattened.data(), sizeWi, hipMemcpyHostToDevice);

        size_t sizeBi = M * sizeof(float);
        hipMemcpy(d_biases[i], biases[i].data(), sizeBi, hipMemcpyHostToDevice);
    }
}

void CUNN::copyWeights(const std::vector<Matrix> weights) {
    assert(weights.size() == numLayers - 1);
    for (int i = 0; i < weights.size(); i++) {
        assert(weights[i].size() == layers[i + 1]);
        assert(weights[i][0].size() == layers[i]);
    }
    this->weights = weights;
}

void CUNN::copyBiases(const std::vector<Vector> biases) {
    assert(biases.size() == numLayers - 1);
    for (int i = 0; i < biases.size(); i++) {
        assert(biases[i].size() == layers[i + 1]);
    }
    this->biases = biases;
}

float CUNN::sigmoid(float x) { return 1.0 / (1.0 + exp(-x)); }
float CUNN::d_sigmoid(float x) {
    float xp = exp(-x);
    return xp / ((1.0 + xp) * (1.0 + xp));
}

Vector& CUNN::forward(const Vector& x, Vector& result) {
    activations[0] = x;
    for (int i = 1; i < numLayers; i++) {
        forwardLayer(weights[i - 1], biases[i - 1], activations[i - 1],
            activations[i]);
    }
    result = activations[numLayers - 1];
    return result;
}

Vector& CUNN::forwardLayer(const Matrix& w, const Vector& b, const Vector& a,
    Vector& result) {
    // return sigmoid(add(multiply(w, a, result), b, result));
    return cu_utility::cuForwardLayer(w, b, a, result);
}

void CUNN::train(const float* d_trainingData, const int* d_trainingLabels, 
    const int M, const int N, const int iterations, const int batchSize,
    float learningRate) {
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < M; i += batchSize) {
            updateFromBatch(d_trainingData+i*N, d_trainingLabels+i, batchSize, N, learningRate);
        }
    }
}

std::vector<float*> CUNN::allocate_like_weights() {
    std::vector<float*> d_Weights;

    // Allocate zeros to accumulate the gradiant over the batch.
    for (int i = 0; i < numLayers - 1; i++) {
        // Allocate memory on the GPU for change in weights
        float* temp_weights;

        size_t weightSize = layers[i + 1] * layers[i] * sizeof(float);

        // Allocate GPU memory
        hipMalloc(&temp_weights, weightSize);

        // Initialize the allocated memory to 0.0 (optional, but often needed)
        hipMemset(temp_weights, 0.0, weightSize);

        // Store pointers in vectors
        d_Weights.push_back(temp_weights);
    }
    return d_Weights;
}
std::vector<float*> CUNN::allocate_like_biases() {
    std::vector<float*> d_Biases;

    // Allocate zeros to accumulate the gradiant over the batch.
    for (int i = 0; i < numLayers - 1; i++) {
        // Allocate memory on the GPU for change in weights and biases
        float* temp_biases;

        size_t biasSize = layers[i + 1] * sizeof(float);

        // Allocate GPU memory
        hipMalloc(&temp_biases, biasSize);

        // Initialize the allocated memory to 0.0 (optional, but often needed)
        hipMemset(temp_biases, 0.0, biasSize);

        // Store pointers in vectors
        d_Biases.push_back(temp_biases);
    }
    return d_Biases;
}
void deallocateVector(std::vector<float*> vec) {
    for (int i = 0; i < vec.size(); i++) {
        hipFree(vec[0]);
    }
}

void CUNN::updateFromBatch(const float* d_batch, const int* d_labels, 
    const int batchSize, const int N, const float learningRate) {

    std::vector<float*> d_ddWeights = allocate_like_weights();
    std::vector<float*> d_ddBiases = allocate_like_biases();

    // calculate individual gradiants and average them together
    for (int i = 0; i < batchSize; i++) {
        std::vector<float*> d_dWeights = allocate_like_weights();
        std::vector<float*> d_dBiases = allocate_like_biases();
        //backwards(d_dWeights, d_dBiases, batch+i*n, labels+i);
        for (int j = 0; j < numLayers-1; j++) {
            //add(d_ddWeights[j], d_dWeights[j], d_ddWeights[j], 1.0 / batchSize);
            //add(d_ddBiases[j], d_dBiases[j], d_ddBiases[j], 1.0 / batchSize);
        }
        deallocateVector(d_dWeights);
        deallocateVector(d_dBiases);
    }
    // update the weights and biases with gradient computed above at the learning rate
    for (int i = 0; i < numLayers-1; i++) {
        //add(d_weights[j], d_ddWeights[j], d_weights[j], -learningRate);
        //add(d_biases[j], d_ddBiases[j], d_biases[j], -learningRate);
    }

    deallocateVector(d_ddWeights);
    deallocateVector(d_ddBiases);
}

void CUNN::backwards(std::vector<Matrix>& dWeights_output,
    std::vector<Vector>& dBiases_output,
    const Vector& testData, int testLabel) {
    activations[0] = testData;
    for (int i = 1; i < numLayers; i++) {
        forwardZ(weights[i - 1], biases[i - 1], activations[i - 1], zs[i]);
        sigmoid(zs[i], activations[i]);
    }
    Vector delta;
    for (int i = 0; i < numLayers - 1; i++) {
        if (i == 0) {
            cost_derivative(activations[activations.size() - 1], testLabel,
                delta);
        }
        else {
            activation_derivative(weights[numLayers - i], zs[numLayers - i],
                delta);
        }
        multiply_elementwise(d_sigmoid(zs[numLayers - 1 - i]), delta,
            dBiases_output[numLayers - 1 - i]);
        outer_product(activations[numLayers - 2 - i],
            dBiases_output[numLayers - 1 - i],
            dWeights_output[numLayers - 1 - i]);
    }
}

void CUNN::cost_derivative(const Vector& last_activation, const int label,
    Vector& result) {
    for (int i = 0; i < 10; i++) {
        if (i == label) {
            result[i] = -1 / last_activation[i];
        }
        else {
            result[i] = 1 / (1 - last_activation[i]);
        }
    }
    return;
}

Matrix& CUNN::outer_product(const Vector& a, const Vector& b, Matrix& result) {
    for (int i = 0; i < a.size(); i++) {
        for (int j = 0; j < b.size(); j++) {
            result[i][j] = a[i] * b[j];
        }
    }
    return result;
}

void CUNN::activation_derivative(const Matrix& weights, Vector& z,
    Vector& previous) {
    d_sigmoid(z);
    multiply_elementwise(z, previous, previous);
    Matrix temp;
    transpose(weights, temp);
    Vector result;
    multiply(temp, previous, result);
    previous = result;
}

void CUNN::transpose(const Matrix& a, Matrix& result) {
    // Get the dimensions of the input matrix 'a'
    int rows = a.size();
    int cols = a[0].size();

    // Resize the result matrix to hold the transpose (cols x rows)
    result.resize(cols);
    for (int i = 0; i < cols; ++i) {
        result[i].resize(rows);
    }

    // Perform the transpose operation
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            result[j][i] = a[i][j];
        }
    }
}

float CUNN::evaluate(const Matrix& testData,
    const std::vector<int>& testLabels) {
    int numCorrect = 0;

    // timing
    auto start = std::chrono::high_resolution_clock::now();

    //Vector result(10, 0);
    //for (int i = 0; i < testData.size(); i++) {
    //    Vector input = testData[i];
    //    Vector output = forward(input, result);

    //    int maxIndex = 0;
    //    float maxVal = 0;
    //    for (int j = 0; j < output.size(); j++) {
    //        if (output[j] > maxVal) {
    //            maxVal = output[j];
    //            maxIndex = j;
    //        }
    //    }
    //    if (maxIndex == testLabels[i]) {
    //        numCorrect++;
    //    }
    //}

    // should be testLabels.size() x 10
    // reserve memory for predictions
	Matrix predictions(testLabels.size(), Vector(10, 0));
	cu_utility::cuForward(d_weights, d_biases, d_activations, layers, testData, predictions);

    for (int i = 0; i < predictions.size(); i++) {
        Vector pred = predictions[i];
        int maxIndex = 0;
        float maxVal = 0;
        for (int j = 0; j < pred.size(); j++) {
            if (pred[j] > maxVal) {
                maxVal = pred[j];
                maxIndex = j;
            }
        }
        if (maxIndex == testLabels[i]) {
            numCorrect++;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    std::cout << "done." << std::endl;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds."
        << std::endl;
    float accuracy = (float)numCorrect / testData.size();
    std::cout << "Train Accuracy: " << accuracy << std::endl;
    return accuracy;
}

Vector& CUNN::forwardZ(const Matrix& w, const Vector& b, const Vector& a,
    Vector& result) {
    return add(multiply(w, a, result), b, result);
}

Vector& CUNN::multiply(const Matrix& w, const Vector& x, Vector& result) {
    cu_utility::cuMatMulVector(w, x, result);
    return result;
}

Vector& CUNN::multiply_elementwise(const Vector& a, const Vector& b,
    Vector& result) {
    for (int i = 0; i < a.size(); i++) {
        result[i] = a[i] * b[i];
    }
    return result;
}

Vector& CUNN::add(const Vector& x, const Vector& b, Vector& result) {
    return cu_utility::cuVectorAdd(x, b, result);
}

Vector& CUNN::add(const Vector& x, const Vector& b, Vector& result,
    const float scale) {
    assert(x.size() == b.size() && x.size() == result.size());

    for (int i = 0; i < x.size(); i++) {
        result[i] = x[i] + b[i] * scale;
    }
    return result;
}

Matrix& CUNN::add(const Matrix& x, const Matrix& b, Matrix& result,
    const float scale) {
    assert(x.size() == b.size() && x.size() == result.size());

    for (int i = 0; i < x.size(); i++) {
        for (int j = 0; j < x[0].size(); j++) {
            result[i][j] = x[i][j] + b[i][j] * scale;
        }
    }
    return result;
}

Vector& CUNN::sigmoid(const Vector& x, Vector& result) {
    result.assign(x.begin(), x.end());
    return cu_utility::cuSigmoid(result);
}

Vector& CUNN::sigmoid(Vector& x) { return cu_utility::cuSigmoid(x); }

Vector& CUNN::d_sigmoid(Vector& x) { return cu_utility::cuDSigmoid(x); }

