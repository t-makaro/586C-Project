#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Kernels
__global__
 void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

// APIs
class cu_utility
{
private:
    /* data */
public:
    cu_utility(/* args */);
    ~cu_utility();
    static std::vector<float>& cuVectorAdd(const std::vector<float> &x, const std::vector<float> &b, std::vector<float> &result);
};

cu_utility::cu_utility(/* args */)
{
}

cu_utility::~cu_utility()
{
}

std::vector<float> &cu_utility::cuVectorAdd(const std::vector<float> &x, const std::vector<float> &b, std::vector<float> &result)
{
    // TODO: insert return statement here
    if(!(x.size() == b.size() && x.size() == result.size())){
        std::cerr << "cuVectorAdd - Size does not match!";
        return result;
    }

    int N = x.size(); // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float *d_x, *d_b, *d_r;
    hipMalloc(&d_x, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_r, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_b, d_r, N);

    // Copy result from device to host
    hipMemcpy(result.data(), d_r, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_r);

    return result;

}

