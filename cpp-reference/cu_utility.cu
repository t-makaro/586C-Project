#include "hip/hip_runtime.h"
#pragma once

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Kernels
__global__
 void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

__device__
float sigmoid(float a){
    return 1.0 / (1.0 + exp(-a));
}

__global__
void sigmoid(float* A,int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = sigmoid(A[i]);
    }
}

__device__
float d_sigmoid(float a){
    float xp = exp(-a);
    return xp / ((1.0 + xp)*(1.0 + xp)); 
}

__global__
void d_sigmoid(float* A, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        A[i] = d_sigmoid(A[i]);
    }
}

// APIs
class cu_utility
{
private:
    /* data */
public:
    cu_utility(/* args */);
    ~cu_utility();
    static std::vector<float>& cuVectorAdd(const std::vector<float> &x, const std::vector<float> &b, std::vector<float> &result);
    static std::vector<float>& cuSigmoid(std::vector<float> &x);
    static std::vector<float> &cu_utility::cuDSigmoid(std::vector<float> &x);
};

cu_utility::cu_utility(/* args */)
{
}

cu_utility::~cu_utility()
{
}

std::vector<float> &cu_utility::cuVectorAdd(const std::vector<float> &x, const std::vector<float> &b, std::vector<float> &result)
{
    if(!(x.size() == b.size() && x.size() == result.size())){
        std::cerr << "cuVectorAdd - Size does not match!";
        return result;
    }

    int N = x.size(); // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float *d_x, *d_b, *d_r;
    hipMalloc(&d_x, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_r, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_b, d_r, N);

    // Copy result from device to host
    hipMemcpy(result.data(), d_r, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_r);

    return result;

}

std::vector<float> &cu_utility::cuSigmoid(std::vector<float> &x){

    int N = x.size(); // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float *d_x;
    hipMalloc(&d_x, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    sigmoid<<<blocksPerGrid, threadsPerBlock>>>(d_x, N);

    // Copy result from device to host
    hipMemcpy(x.data(), d_x, size, hipMemcpyDeviceToHost);

    hipFree(d_x);

    return x;
}

std::vector<float> &cu_utility::cuDSigmoid(std::vector<float> &x){
    int N = x.size(); // Size of vectors
    size_t size = N * sizeof(float);

    // Allocate device memory
    float *d_x;
    hipMalloc(&d_x, size);

    // Copy data from host to device
    hipMemcpy(d_x, x.data(), size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    d_sigmoid<<<blocksPerGrid, threadsPerBlock>>>(d_x, N);

    // Copy result from device to host
    hipMemcpy(x.data(), d_x, size, hipMemcpyDeviceToHost);

    hipFree(d_x);

    return x;
}

